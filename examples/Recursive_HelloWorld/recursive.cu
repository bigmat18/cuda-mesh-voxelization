
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void nestedHelloWorld(const int iSize, int iDepth)
{
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d block %d\n", iDepth, tid, blockIdx.x);
    // condition to stop recursive execution
    if (iSize == 1) {
        return;
    }
    // reduce block size to half
    int nthreads = iSize >> 1;
    // thread 0 launches child grid recursively
    if (tid == 0 && nthreads > 0) {
        nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth);
        printf("-------> nested execution depth: %d\n", iDepth);
    }
}
