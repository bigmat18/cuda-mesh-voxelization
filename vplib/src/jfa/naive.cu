#include "hip/hip_runtime.h"
#include "grid/grid.h"
#include "grid/voxels_grid.h"
#include "mesh/mesh.h"
#include <cmath>
#include <iostream>
#include <jfa/jfa.h>
#include <limits>

namespace JFA {

template <typename T>
__global__ void InizializationNaive(const VoxelsGrid<T, true> grid, Grid<float> sdf, Grid<Position> positions) 
{
    const int voxelIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(voxelIndex >= grid.Size())
        return;

    const int voxelZ = voxelIndex / (grid.VoxelsPerSide() * grid.VoxelsPerSide());
    const int voxelY = (voxelIndex % (grid.VoxelsPerSide() * grid.VoxelsPerSide())) / grid.VoxelsPerSide();
    const int voxelX = voxelIndex % grid.VoxelsPerSide();

    if(!grid.Voxel(voxelX, voxelY, voxelZ))
        return;


    bool found = false;
    Position pos = Position(grid.OriginX() + (voxelX * grid.VoxelSize()),
                            grid.OriginY() + (voxelY * grid.VoxelSize()),
                            grid.OriginZ() + (voxelZ * grid.VoxelSize()));

    for(int z = -1; z <= 1; z++) {
        for(int y = -1; y <= 1; y++) {
            for(int x = -1; x <= 1; x++) {
                if(x == 0 && y == 0 && z == 0)
                    continue;

                int nx = voxelX + x;
                int ny = voxelY + y;
                int nz = voxelZ + z;

                bool isBorder = nx < 0 || nx >= grid.VoxelsPerSide() || 
                                ny < 0 || ny >= grid.VoxelsPerSide() || 
                                nz < 0 || nz >= grid.VoxelsPerSide();

                if(isBorder || !grid.Voxel(nx, ny, nz))
                    found = true;
            }
        }
    }
    if(found) {
        sdf(voxelX, voxelY, voxelZ) = 0.0f;
        positions(voxelX, voxelY, voxelZ) = pos;
    } else {
        sdf(voxelX, voxelY, voxelZ) = INFINITY;
    }
}

template <typename T>
__global__ void ProcessingNaive(const VoxelsGrid<T, true> grid, 
                                Grid<float> sdf, Grid<Position> positions) {

    const int voxelIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(voxelIndex >= grid.Size())
        return;

    const int voxelZ = voxelIndex / (grid.VoxelsPerSide() * grid.VoxelsPerSide());
    const int voxelY = (voxelIndex % (grid.VoxelsPerSide() * grid.VoxelsPerSide())) / grid.VoxelsPerSide();
    const int voxelX = voxelIndex % grid.VoxelsPerSide();

    Position voxelPos = Position(grid.OriginX() + (voxelX * grid.VoxelSize()),
                                 grid.OriginY() + (voxelY * grid.VoxelSize()),
                                 grid.OriginZ() + (voxelZ * grid.VoxelSize()));

    for(int k = grid.VoxelsPerSide() / 2; k >= 1; k/=2)
    {
        bool findNewBest = false;
        float bestDistance = sdf(voxelX, voxelY, voxelZ);
        Position bestPosition;
        for(int z = -1; z <= 1; z++) {
            for(int y = -1; y <= 1; y++) {
                for(int x = -1; x <= 1; x++) {
                    if(x == 0 && y == 0 && z == 0)
                        continue;

                    int nx = voxelX + (x * k);
                    int ny = voxelY + (y * k);
                    int nz = voxelZ + (z * k);

                    if(nx < 0 || nx >= grid.VoxelsPerSide() ||
                       ny < 0 || ny >= grid.VoxelsPerSide() ||
                       nz < 0 || nz >= grid.VoxelsPerSide())
                        continue;

                    float seed = sdf(nx, ny, nz);
                    if(fabs(seed) < INFINITY) {
                        Position seedPos = positions(nx, ny, nz);

                        float distance = CalculateDistance(voxelPos, seedPos);
                        if(distance < fabs(bestDistance)) {
                            findNewBest = true;
                            bestDistance = copysignf(distance, bestDistance);
                            bestPosition = seedPos;
                        }
                    }
                }
            }
        }

        __syncthreads();
        if (findNewBest) {
            sdf(voxelX, voxelY, voxelZ) = bestDistance;
            positions(voxelX, voxelY, voxelZ) = bestPosition;
        }
        __syncthreads();
    }
}

template <Types type, typename T>
void Compute<Types::NAIVE, T>(HostVoxelsGrid<T>& grid, HostGrid<float>& sdf)
{ 
    PROFILING_SCOPE("NaiveJFA");

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const size_t numVoxels = grid.View().Size();
    const size_t blockSize = NextPow2(numVoxels, prop.maxThreadsDim[0] / 2);
    const size_t gridSize = (numVoxels + blockSize - 1) / blockSize;

    DeviceVoxelsGrid<T> devGrid;
    DeviceGrid<float> devSDF;
    DeviceGrid<Position> devPositions;
    {
        PROFILING_SCOPE("NaiveJFA::Memory");
        devGrid = DeviceVoxelsGrid<T>(grid);
        devSDF = DeviceGrid<float>(sdf);
        devPositions = DeviceGrid<Position>(grid.View().VoxelsPerSide());
    }

    {
        PROFILING_SCOPE("NaiveJFA::Inizialization");
        InizializationNaive<T><<< gridSize, blockSize >>>(
            devGrid.View(), devSDF.View(), devPositions.View()
        );
        gpuAssert(hipPeekAtLastError());
        hipDeviceSynchronize();
    }
    
    {
        PROFILING_SCOPE("NaiveJFA::Processing");

        ProcessingNaive<T><<< gridSize, blockSize >>>(
            devGrid.View(), devSDF.View(), devPositions.View()
        );
        gpuAssert(hipPeekAtLastError()); 
        hipDeviceSynchronize();
    }

    {
        PROFILING_SCOPE("NaiveJFA::Memory");
        sdf = HostGrid<float>(devSDF);
    }
};


template void Compute<Types::NAIVE, uint32_t>
(HostVoxelsGrid<uint32_t>&, HostGrid<float>&);

template void Compute<Types::NAIVE, uint64_t>
(HostVoxelsGrid<uint64_t>&, HostGrid<float>&);


template __global__ void InizializationNaive<uint32_t>
(const VoxelsGrid<uint32_t, true>, Grid<float>, Grid<Position>);

template __global__ void InizializationNaive<uint64_t>
(const VoxelsGrid<uint64_t, true>, Grid<float>, Grid<Position>);


template __global__ void ProcessingNaive<uint32_t>
(const VoxelsGrid<uint32_t, true>, Grid<float>, Grid<Position>);

template __global__ void ProcessingNaive<uint64_t>
(const VoxelsGrid<uint64_t, true>, Grid<float>, Grid<Position>);

}
